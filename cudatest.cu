#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>


//for cublas function
#include<hipblas.h>

// simple kernel function that adds two vectors
__global__ void vect_add(float *a, float *b, int N)
{
   int idx = threadIdx.x;
   if (idx<N) a[idx] = a[idx] + b[idx];
}

// function called from main fortran program
extern "C" void kernel_wrapper_(float *a, float *b, int *Np)
{
   float  *a_d, *b_d;  // declare GPU vector copies
   
   int blocks = 64;     // uses 1 block of
   int N = *Np;        // N threads on GPU

N = N/blocks;

   // Allocate memory on GPU
   hipMalloc( (void **)&a_d, sizeof(float) * N );
   hipMalloc( (void **)&b_d, sizeof(float) * N );

   // copy vectors from CPU to GPU
   hipMemcpy( a_d, a, sizeof(float) * N, hipMemcpyHostToDevice );
   hipMemcpy( b_d, b, sizeof(float) * N, hipMemcpyHostToDevice );

   // call function on GPU
   //vect_add<<< blocks, N >>>( a_d, b_d, N);
   vect_add<<< blocks, N >>>( a_d, b_d, N);

   // copy vectors back from GPU to CPU
   hipMemcpy( a, a_d, sizeof(float) * N, hipMemcpyDeviceToHost );
   //cudaMemcpy( b, b_d, sizeof(float) * N, cudaMemcpyDeviceToHost );

   // free GPU memory
   hipFree(a_d);
   hipFree(b_d);
   return;
}

/*
extern "C" void kernel_wrapper_cublas_saxpy(float* A, float* B, int size_A)
{

checkCudaErrors(cudaMalloc((void **) &d_A, mem_size_A));

 cublasHandle_t handle;
 checkCudaErrors(cublasCreate(&handle));
 float alpha = 1.0;
 int incx =1;
 int incy =1;
 checkCudaErrors(cublasSaxpy(handle, size_A, &alpha, d_A, incx, d_B, incy )); 

}
*/