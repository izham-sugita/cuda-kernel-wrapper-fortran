#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>


// simple kernel function that adds two vectors
// originally used for demonstration
__global__ void vect_add(float *a, float *b, int N)
{
   int idx = threadIdx.x;
   if (idx<N) a[idx] = a[idx] + b[idx];
}

__global__ void vectorAdd(float *a, float *b, int N)
{
int i = blockDim.x*blockIdx.x + threadIdx.x;
if(i < N) a[i] = a[i] + b[i];
}

//function to call from FORTRAN
extern "C" void vectoraddwrapper_( float *a, float *b, int *Np)
{
hipError_t err = hipSuccess;
int N = *Np;       // number of elements

size_t size = N*sizeof(float);

float *d_a = NULL;
err = hipMalloc((void **)&d_a, size);
//error check
if(err != hipSuccess)
{
fprintf(stderr,"Failed to allocate memory for vector A! \n");
exit(EXIT_FAILURE);
}

float *d_b = NULL;
err = hipMalloc((void **)&d_b, size);
//error check
if(err != hipSuccess)
{
fprintf(stderr,"Failed to allocate memory for vector B! \n");
exit(EXIT_FAILURE);
}

//copying value from host
err = hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
err = hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

//setting up computation kernel
int threadsPerBlock = 1; //require testing
int blocksPerGrid =(N + threadsPerBlock - 1) / threadsPerBlock;
vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, N);

//copy result
err = hipMemcpy(a, d_a, size, hipMemcpyDeviceToHost);
//error check
if(err != hipSuccess)
{
fprintf(stderr,"Failed to copy result for vector A! \n");
exit(EXIT_FAILURE);
}

hipFree(d_a);
hipFree(d_b);
printf("Test passed!\n");

}
