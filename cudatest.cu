#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>


//for cublas function
#include<hipblas.h>

// simple kernel function that adds two vectors
__global__ void vect_add(float *a, float *b, int N)
//__global__ void vect_add(float *a, float *b, ulong N)
{
   int idx = threadIdx.x;
   //ulong idx = threadIdx.x;
   if (idx<N) a[idx] = a[idx] + b[idx];
}

__global__ void vectorAdd(float *a, float *b, int N)
{
int i = blockDim.x*blockIdx.x + threadIdx.x;
if(i < N) a[i] = a[i] + b[i];
}

//function to call from FORTRAN
extern "C" void vectoraddwrapper_( float *a, float *b, int *Np)
{
hipError_t err = hipSuccess;
int N = *Np;       // number of elements

size_t size = N*sizeof(float);

float *d_a = NULL;
err = hipMalloc((void **)&d_a, size);
//error check
if(err != hipSuccess)
{
fprintf(stderr,"Failed to allocate memory for vector A! \n");
exit(EXIT_FAILURE);
}

float *d_b = NULL;
err = hipMalloc((void **)&d_b, size);
//error check
if(err != hipSuccess)
{
fprintf(stderr,"Failed to allocate memory for vector B! \n");
exit(EXIT_FAILURE);
}

//copying value from host
err = hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
err = hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

//setting up computation kernel
int threadsPerBlock = 1; //require testing
int blocksPerGrid =(N + threadsPerBlock - 1) / threadsPerBlock;
vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, N);

//copy result
err = hipMemcpy(a, d_a, size, hipMemcpyDeviceToHost);
//error check
if(err != hipSuccess)
{
fprintf(stderr,"Failed to copy result for vector A! \n");
exit(EXIT_FAILURE);
}

hipFree(d_a);
hipFree(d_b);
printf("Test passed!\n");

}

// function called from main fortran program
/*
extern "C" void kernel_wrapper_(float *a, float *b, int *Np)
{
   float  *a_d, *b_d;  // declare GPU vector copies
   
   int blocks = 1;     // uses 1 block of
   int N = *Np;        // N threads on GPU
   //N = N/blocks;

//ulong blocks = 2;
//ulong N = *Np;
//N = N/(blocks+1);

   // Allocate memory on GPU
   cudaMalloc( (void **)&a_d, sizeof(float) * N );
   cudaMalloc( (void **)&b_d, sizeof(float) * N );

   // copy vectors from CPU to GPU
   cudaMemcpy( a_d, a, sizeof(float) * N, cudaMemcpyHostToDevice );
   cudaMemcpy( b_d, b, sizeof(float) * N, cudaMemcpyHostToDevice );

   // call function on GPU
   //vect_add<<< blocks, N >>>( a_d, b_d, N);
   vect_add<<< blocks, N >>>( a_d, b_d, N);

   // copy vectors back from GPU to CPU
   cudaMemcpy( a, a_d, sizeof(float) * N, cudaMemcpyDeviceToHost );
   //cudaMemcpy( b, b_d, sizeof(float) * N, cudaMemcpyDeviceToHost );

   // free GPU memory
   cudaFree(a_d);
   cudaFree(b_d);
   return;
}*/


/*
extern "C" void kernel_wrapper_cublas_saxpy(float* A, float* B, int size_A)
{

checkCudaErrors(cudaMalloc((void **) &d_A, mem_size_A));

 cublasHandle_t handle;
 checkCudaErrors(cublasCreate(&handle));
 float alpha = 1.0;
 int incx =1;
 int incy =1;
 checkCudaErrors(cublasSaxpy(handle, size_A, &alpha, d_A, incx, d_B, incy )); 

}
*/